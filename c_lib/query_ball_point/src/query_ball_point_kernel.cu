#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "query_ball_point_kernel.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *__restrict__ xyz1, const float *__restrict__ xyz2, long *__restrict__ idx, int *__restrict__ pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n*3*batch_index;
    xyz2 += m*3*batch_index;
    idx += m*nsample*batch_index;
    pts_cnt += m*batch_index; // counting how many unique points selected in local region

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        int cnt = 0;
        for (int k=0;k<n;++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball
            float x2=xyz2[j*3+0];
            float y2=xyz2[j*3+1];
            float z2=xyz2[j*3+2];
            float x1=xyz1[k*3+0];
            float y1=xyz1[k*3+1];
            float z1=xyz1[k*3+2];
            float d=max(sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)),1e-20f);
            if (d<radius) {
                if (cnt==0) { // set ALL indices to k, s.t. if there are less points in ball than nsample, we still have valid (repeating) indices
                    for (int l=0;l<nsample;++l)
                        idx[j*nsample+l] = k;
                }
                idx[j*nsample+cnt] = k;
                cnt+=1;
            }
        }
        pts_cnt[j] = cnt;
    }
}

//require 32*n working space
int queryBallPointLauncher(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, long *idx, int *pts_cnt, hipStream_t stream) {
    hipError_t err;
    query_ball_point_gpu<<<b,256, 0, stream>>>(b,n,m,radius,nsample,xyz1,xyz2,idx,pts_cnt);
    err = hipGetLastError();
    if(hipSuccess != err)
    {
        fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }

    return 1;
}
#ifdef __cplusplus
}
#endif